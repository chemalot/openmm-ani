#include "hip/hip_runtime.h"
/* -------------------------------------------------------------------------- *
 * The MIT License
 *
 * SPDX short identifier: MIT
 *
 * Copyright 2019 Genentech Inc. South San Francisco
 *
 * Permission is hereby granted, free of charge, to any person obtaining a
 * copy of this software and associated documentation files (the "Software"),
 * to deal in the Software without restriction, including without limitation
 * the rights to use, copy, modify, merge, publish, distribute, sublicense,
 * and/or sell copies of the Software, and to permit persons to whom the
 * Software is furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included
 * in all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS
 * OR IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER
 * DEALINGS IN THE SOFTWARE.
 * -------------------------------------------------------------------------- */
/* -------------------------------------------------------------------------- *
 * Portions of this software were derived from code originally developed
 * by Peter Eastman and copyrighted by Stanford University and the Authors
 * -------------------------------------------------------------------------- */

extern "C" __global__
void addForces(const FORCES_TYPE* __restrict__ forces, long long* __restrict__ forceBuffers, int* __restrict__ atomIndex, int numAtoms, int paddedNumAtoms) {
    for (int atom = blockIdx.x*blockDim.x+threadIdx.x; atom < numAtoms; atom += blockDim.x*gridDim.x) {
        int index = atomIndex[atom];
        forceBuffers[atom] += (long long) (forces[3*index]*0x100000000);
        forceBuffers[atom+paddedNumAtoms] += (long long) (forces[3*index+1]*0x100000000);
        forceBuffers[atom+2*paddedNumAtoms] += (long long) (forces[3*index+2]*0x100000000);
    }
}

